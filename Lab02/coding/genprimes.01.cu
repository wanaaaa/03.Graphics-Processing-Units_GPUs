#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 
#include <string.h>

#include <iostream>
using namespace std;

__global__ void removeMultiple(int * playground, int N, int startNum);

int main(int argc, char * argv[]) {
   double time_taken;
   clock_t start, end;

   int N = (unsigned int) atoi(argv[1]);
   int startNum = 2;
   size_t groundSize = (N-1)*sizeof(int);

   int * playground = (int *)calloc(N-1, sizeof(int));
   for(int i = 0; i < N-1; i ++ ) {
      playground[i] = i + 2;
   }

   int *d_playground;
   hipMalloc((void **)&d_playground, groundSize);
   hipMemcpy(d_playground, playground, groundSize, hipMemcpyHostToDevice);

   int threadNum = 256;
   int blockNum = (N -1 + threadNum -1)/threadNum;

   int wI = 0;
   start = clock();

   while(startNum  < (N+1)/2) {
      if(playground[startNum-2] != -1)
         ////////////////////////////////////////
         removeMultiple<<< blockNum, threadNum >>>(d_playground, N, startNum);
         ////////////////////////////////////////

      startNum ++;
      wI ++;
   }
   end = clock();

   char str[8000];
   strcpy(str, "");

   int *h_playgroundResult = (int *)malloc(groundSize);
   hipMemcpy(h_playgroundResult, d_playground, groundSize, hipMemcpyDeviceToHost);

   
   int numPrime = 0;
   for(int i = 0; i < N-1; i++) {
      if(h_playgroundResult[i] != -1) {
         char new_string[6];
         int number=playground[i];
         sprintf(new_string ,"%d" , number);       
         strcat(str, new_string);
         strcat(str, " ");
         // cout<<"prime: i->"<<i+2<<" value->"<<h_playgroundResult[i]<<endl;
         numPrime ++;
      }
   }


   cout<<"the number of prime is "<<numPrime<<endl;

   hipFree(d_playground);

   time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
   printf("Time taken for %s is %lf\n", "GPU", time_taken);

   return 0;
}

__global__ void removeMultiple(int * playground, int N, int startNum) {
   int ix = threadIdx.x + blockDim.x*blockIdx.x;

   if((ix<N-1) && (ix > startNum -1) ) {
      if((playground[ix] % startNum == 0)&& (playground[ix] != -1 ) ) {
         playground[ix] = -1;        
      }
   }
}

