/*
 *  Please write your name and net ID below
 *  
 *  Last name:
 *  First name:
 *  Net ID: 
 * 
 */


/* 
 * This file contains the code for doing the heat distribution problem. 
 * You do not need to modify anything except starting  gpu_heat_dist() at the bottom
 * of this file.
 * In gpu_heat_dist() you can organize your data structure and the call to your
 * kernel(s) that you need to write too. 
 * 
 * You compile with:
 * 		nvcc -o heatdist heatdist.cu   
 */

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h> 

/* To index element (i,j) of a 2D array stored as 1D */
#define index(i, j, N)  ((i)*(N)) + (j)

#include <iostream>
using namespace std;
/*****************************************************************/

// Function declarations: Feel free to add any functions you want.
void  seq_heat_dist(float *, unsigned int, unsigned int);
void  gpu_heat_dist(float *, unsigned int, unsigned int);


/*****************************************************************/

int main(int argc, char * argv[])
{
  // printf("asdfasdfasdfasfd\n");
  // std::cout<<"adfasdf============================================"<<std::endl;
  unsigned int N; /* Dimention of NxN matrix */
  int type_of_device = 0; // CPU or GPU
  int iterations = 0;
  int i;
  
  /* The 2D array of points will be treated as 1D array of NxN elements */
  float * playground; 
  
  // to measure time taken by a specific part of the code 
  double time_taken;
  clock_t start, end;
  
  if(argc != 4)
  {
    fprintf(stderr, "usage: heatdist num  iterations  who\n");
    fprintf(stderr, "num = dimension of the square matrix (50 and up)\n");
    fprintf(stderr, "iterations = number of iterations till stopping (1 and up)\n");
    fprintf(stderr, "who = 0: sequential code on CPU, 1: GPU execution\n");
    exit(1);
  }
  
  type_of_device = atoi(argv[3]);
  N = (unsigned int) atoi(argv[1]);
  iterations = (unsigned int) atoi(argv[2]);
 
  
  /* Dynamically allocate NxN array of floats */
  playground = (float *)calloc(N*N, sizeof(float));
  if( !playground )
  {
   fprintf(stderr, " Cannot allocate the %u x %u array\n", N, N);
   exit(1);
  }
  
  /* Initialize it: calloc already initalized everything to 0 */
  // Edge elements to 80F
  for(i = 0; i < N; i++)
    playground[index(0,i,N)] = 80;
    
  for(i = 0; i < N; i++)
    playground[index(i,0,N)] = 80;
  
  for(i = 0; i < N; i++)
    playground[index(i,N-1, N)] = 80;
  
  for(i = 0; i < N; i++)
    playground[index(N-1,i,N)] = 80;
  
  // from (0,10) to (0,30) inclusive are 150F
  for(i = 10; i <= 30; i++)
    playground[index(0,i,N)] = 150;
  
  
  if( !type_of_device ) // The CPU sequential version
  {  
    start = clock();
    seq_heat_dist(playground, N, iterations);
    end = clock();
  }
  else  // The GPU version
  {
     start = clock();
     gpu_heat_dist(playground, N, iterations); 
     end = clock();    
  }
  
  
  time_taken = ((double)(end - start))/ CLOCKS_PER_SEC;
  
  printf("Time taken for %s is %lf\n", type_of_device == 0? "CPU" : "GPU", time_taken);
  
  free(playground);
  
  return 0;

}


/*****************  The CPU sequential version (DO NOT CHANGE THAT) **************/
void  seq_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{
  // Loop indices
  int i, j, k;
  int upper = N-1;
  
  // number of bytes to be copied between array temp and array playground
  unsigned int num_bytes = 0;
  
  float * temp; 
  /* Dynamically allocate another array for temp values */
  /* Dynamically allocate NxN array of floats */
  temp = (float *)calloc(N*N, sizeof(float));
  if( !temp )
  {
   fprintf(stderr, " Cannot allocate temp %u x %u array\n", N, N);
   exit(1);
  }
  
  num_bytes = N*N*sizeof(float);
  
  /* Copy initial array in temp */
  memcpy((void *)temp, (void *) playground, num_bytes);
  
  for( k = 0; k < iterations; k++)
  {
    /* Calculate new values and store them in temp */
    for(i = 1; i < upper; i++)
      for(j = 1; j < upper; j++)
            temp[index(i,j,N)] = (playground[index(i-1,j,N)] + 
            playground[index(i+1,j,N)] + 
            playground[index(i,j-1,N)] + 
            playground[index(i,j+1,N)])/4.0;
  
            
              
    /* Move new values into old values */ 
    memcpy((void *)playground, (void *) temp, num_bytes);
  }
  
}

/***************** The GPU version: Write your code here *********************/
/* This function can call one or more kenels *********************************/
// __global__ void testLoop(float * tempGround, float * playground, unsigned int N, unsigned int iterations);
__global__ void testLoop(float * tempGround, float * playground, int intN);

void  gpu_heat_dist(float * playground, unsigned int N, unsigned int iterations)
{
   cout<<"~~~in gpu_heat_dist"<<endl;
   int numElements = N*N;
   size_t groundSize = numElements * sizeof(float);

   float *h_temp = (float *) malloc(groundSize);

   float *d_temp, *d_playground;

   float *h_playgroundResult = (float *)malloc(groundSize);

   hipMalloc((void **)&d_temp, groundSize);
   hipMalloc((void **)&d_playground, groundSize);

   hipMemcpy(d_temp, h_temp, groundSize, hipMemcpyHostToDevice);
   hipMemcpy(d_playground, playground, groundSize, hipMemcpyHostToDevice);

   ///////////////////////////////////
   int threadNum = 256;
   int blockNum = (N + threadNum -1)/threadNum;

   // testLoop<<<blockNum, threadNum >>>(d_temp, numElements);
   testLoop<<<blockNum, threadNum >>>(d_temp, d_playground, N);

   hipMemcpy(h_playgroundResult, d_playground, groundSize,  hipMemcpyDeviceToHost);

   for(int i = 0; i < N*N; i++) {
      cout<<"i-> "<<i/N<<" j->"<<i % N<< " value->"<<  h_playgroundResult[i]<<endl;
   }

   cout<<"N is "<<N<<endl;
   hipFree(d_temp); hipFree(d_playground);

}

__global__ void testLoop(float * tempGround, float * playground, int intN) {
   int ix = threadIdx.x + blockDim.x*blockIdx.x;
   // int iy = threadIdx.y + blockDim.y*blockIdx.y;

   if(ix < intN ) {
      for(int i = 1; i < intN -1 ; i++) {
        // tempGround[ix*intN + i] = (float) ix+ 0.777;  
        if ((ix > 0) && (ix < intN -1))   {
          playground[index(ix, i, intN)] = (float) ix+ 0.123;         
        }    
      }
   }

  // tempGround[ix*10+iy] = (float) iy+ 0.777; 

}
